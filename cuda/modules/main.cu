#include "hip/hip_runtime.h"
#include "../includes/bindings.h"
#include "../includes/libraries/glm/glm.hpp"
#include "../includes/ray_marching.cu"
#include "../includes/rendering.cu"
#include "../includes/utils.h"

using namespace glm;

#define RELATIVIZE_STEP_COUNT false

// coordinate system conversion

__device__ vec2 texture_to_ndc(uvec2 p, vec2 texture_size) {
    return ((vec2) p + vec2(0.5f, 0.5f)) / texture_size;
}

__device__ uvec2 ndc_to_texture(vec2 p, vec2 texture_size) {
    return uvec2(round((p * texture_size) - vec2(0.5f, 0.5f)));
}

__device__ vec2 ndc_to_camera(vec2 p, vec2 render_screen_size) {
    return {(2 * p.x - 1) * (render_screen_size.x / render_screen_size.y), 1 - 2 * p.y};
}

__device__ vec3 camera_to_ray(vec2 p, CameraBuffer CAMERA) {
    float fov_fac = tan(CAMERA.fov / 2);
    return normalize(
        vec3(CAMERA.forward[0], CAMERA.forward[1], CAMERA.forward[2])
        + p.y * fov_fac * vec3(CAMERA.up[0], CAMERA.up[1], CAMERA.up[2])
        + p.x * fov_fac * vec3(CAMERA.right[0], CAMERA.right[1], CAMERA.right[2])
    );
}

// scene

__shared__ SdRuntimeScene runtime_scene;

__device__ auto make_sd_scene(
    GlobalsBuffer &globals,
    CameraBuffer &camera
) {
    return [globals](vec3 p){
        float sd = sd_box(p, vec3(-30.0f, 0.0f, 0.0f), vec3(1.0f, 2.0f, 10.0f));

        //p.x = wrap(p.x, -40.0f, 40.0f);
        //sd = min(sd_mandelbulb((p + vec3(0.0f, 0.0f, 15.0f)) / 20.0f, globals.time) * 20.0f, sd);

        for (int i = 0; i < runtime_scene.sphere_count; i++) {
            sd = min(
                sd,
                length(p - from_array(runtime_scene.spheres[i].translation)) - runtime_scene.spheres[i].radius
            );
        }

        return sd;
    };
}

// ray-marching

#include <hip/hip_runtime.h>

#ifndef DISABLE_CONE_MARCH
extern "C" __global__ void compute_compressed_depth(
     unsigned int level,
     RenderDataTexture render_data_texture,
     ConeMarchTextures cm_textures,
     GlobalsBuffer globals,
     CameraBuffer camera,
     SdRuntimeScene runtime_scene_param
) {
    runtime_scene.sphere_count = runtime_scene_param.sphere_count;

    int perThread = runtime_scene.sphere_count / blockDim.x;
    for (int i = 0; i < perThread; i++) {
        runtime_scene.spheres[threadIdx.x * perThread + i] = runtime_scene_param.spheres[threadIdx.x * perThread + i];
    }

    __syncthreads();

    u32 id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id > cm_textures.textures[level].size[0] * cm_textures.textures[level].size[1]) {
        return;
    }

    uvec2 cm_texture_coord = uvec2(id % cm_textures.textures[level].size[0], id / cm_textures.textures[level].size[0]);
    vec2 ndc_coord = texture_to_ndc(cm_texture_coord, {cm_textures.textures[level].size[0], cm_textures.textures[level].size[1] });
    vec2 cam_coord = ndc_to_camera(ndc_coord, { cm_textures.textures[level].size[0], cm_textures.textures[level].size[1] });
    Ray ray { { camera.position[0], camera.position[1], camera.position[2] }, camera_to_ray(cam_coord, camera) };

    float aspect_ratio = (float) render_data_texture.size[0] / (float) render_data_texture.size[1];
    float fov_fac = tan(camera.fov / 2);
    float cone_radius = length(vec2(
        (2.0f * aspect_ratio * fov_fac) / (float) cm_textures.textures[level].size[0],
        (2.0f * fov_fac) / (float) cm_textures.textures[level].size[1]
    ));

    ConeMarchTextureValue entry { 0.0f, 0, Collision };
    if (level > 0) {
        uvec2 lower_cm_texture_coord = ndc_to_texture(
        ndc_coord,
        { (float) cm_textures.textures[level - 1].size[0], (float) cm_textures.textures[level - 1].size[1] }
        );

        entry = cm_textures.textures[level - 1].texture[
            lower_cm_texture_coord.x + cm_textures.textures[level - 1].size[0] * lower_cm_texture_coord.y
        ];
    }

    RayMarchHit hit = ray_march<true>(make_sd_scene(globals, camera), ray, entry, cone_radius);

    if (RELATIVIZE_STEP_COUNT) {
        float compression_factor = (float) (render_data_texture.size[0] * render_data_texture.size[1]) / (float) (cm_textures.textures[level].size[0] * cm_textures.textures[level].size[1]);
        hit.steps = (int) ceil((float) hit.steps / compression_factor);
    }

    cm_textures.textures[level].texture[id] = ConeMarchTextureValue { hit.depth, hit.steps, hit.outcome };
}
#endif

extern "C" __global__ void compute_render(
    RenderDataTexture render_data_texture,
    ConeMarchTextures cm_textures,
    GlobalsBuffer globals,
    CameraBuffer camera,
    SdRuntimeScene runtime_scene_param,
    bool compression_enabled
) {
    runtime_scene.sphere_count = runtime_scene_param.sphere_count;

    int perThread = runtime_scene.sphere_count / blockDim.x;
    for (int i = 0; i < perThread; i++) {
        runtime_scene.spheres[threadIdx.x * perThread + i] = runtime_scene_param.spheres[threadIdx.x * perThread + i];
    }

    __syncthreads();

    u32 id = blockIdx.x * blockDim.x + threadIdx.x;
    uvec2 texture_coord = uvec2(id % render_data_texture.size[0], id / render_data_texture.size[0]);
    vec2 ndc_coord = texture_to_ndc(texture_coord, { render_data_texture.size[0], render_data_texture.size[1] });
    vec2 cam_coord = ndc_to_camera(ndc_coord, { render_data_texture.size[0], render_data_texture.size[1] });
    Ray ray { { camera.position[0], camera.position[1], camera.position[2] }, camera_to_ray(cam_coord, camera) };

    #ifndef DISABLE_CONE_MARCH
        ConeMarchTextureValue entry = { 0.0f, 0, Collision };

        if (compression_enabled) {
            uvec2 cm_texture_coord = ndc_to_texture(
                    ndc_coord,
                    {(float) cm_textures.textures[CONE_MARCH_LEVELS - 1].size[0],
                     (float) cm_textures.textures[CONE_MARCH_LEVELS - 1].size[1]}
            );

            entry = cm_textures.textures[CONE_MARCH_LEVELS - 1].texture[
                    cm_texture_coord.x + cm_textures.textures[CONE_MARCH_LEVELS - 1].size[0] * cm_texture_coord.y
            ];
        }
    #else
        ConeMarchTextureValue entry = { 0.0f, 0, Collision };
    #endif

    RayMarchHit hit = ray_march<false>(make_sd_scene(globals, camera), ray, entry);

    render_data_texture.texture[id] = {
            hit.depth, (float) hit.steps, hit.outcome, { 1.0f, 1.0f, 1.0f }, 1.0f
    };
}

extern "C" __global__ void compute_render_finalize(
    Texture render_texture,
    RenderDataTexture render_data_texture,
    GlobalsBuffer globals
) {
    u32 id = blockIdx.x * blockDim.x + threadIdx.x;
    uvec2 texture_coord = uvec2(id % render_data_texture.size[0], id / render_data_texture.size[0]);

    vec3 color = clamp(
            vec3(render_data_texture.texture[id].steps * 0.001f),
            0.0f, 1.0f
    );

    unsigned int rgba = ((unsigned int)(255.0f * color.x) & 0xff) |
                        (((unsigned int)(255.0f * color.y) & 0xff) << 8) |
                        (((unsigned int)(255.0f * color.z) & 0xff) << 16) |
                        (255 << 24);

    render_texture.texture[id] = rgba;
}
