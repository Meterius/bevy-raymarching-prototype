#include "hip/hip_runtime.h"
#include "../includes/bindings.h"
#include "../includes/libraries/glm/glm.hpp"
#include "./signed_distance.cu"
#include "./ray_marching.cu"

using namespace glm;

// coordinate system conversion

__device__ vec2 texture_to_ndc(uvec2 p, vec2 texture_size) {
    return ((vec2) p + vec2(0.5f, 0.5f)) / texture_size;
}

__device__ vec2 ndc_to_camera(vec2 p, vec2 render_screen_size) {
    return {(2 * p.x - 1) * (render_screen_size.x / render_screen_size.y), 1 - 2 * p.y};
}

__device__ vec3 camera_to_ray(vec2 p, CameraBuffer CAMERA) {
    float fov_fac = tan(CAMERA.fov / 2);
    return normalize(
        vec3(CAMERA.forward[0], CAMERA.forward[1], CAMERA.forward[2])
        + p.y * fov_fac * vec3(CAMERA.up[0], CAMERA.up[1], CAMERA.up[2])
        + p.x * fov_fac * vec3(CAMERA.right[0], CAMERA.right[1], CAMERA.right[2])
    );
}

// ray-marching

__forceinline__ __device__ vec3 render_ray(Ray ray, float time, DepthTextureEntry starting) {
    RayMarchHit hit = ray_march(ray, time, starting);
    return vec3(hit.depth * 0.001f, f32(hit.outcome == StepLimit), (float) hit.steps * 0.001f);
}


extern "C" __global__ void render_depth(char *render_texture, DepthTexture depth_texture, GlobalsBuffer globals, CameraBuffer camera) {
    u32 id = blockIdx.x * blockDim.x + threadIdx.x;
    uvec2 depth_texture_coord = uvec2(id % depth_texture.size[0], id / depth_texture.size[0]);
    vec2 ndc_coord = texture_to_ndc(depth_texture_coord, {depth_texture.size[0], depth_texture.size[1] });
    vec2 cam_coord = ndc_to_camera(ndc_coord, { depth_texture.size[0], depth_texture.size[1] });
    Ray ray { { camera.position[0], camera.position[1], camera.position[2] }, camera_to_ray(cam_coord, camera) };

    float aspect_ratio = (float) globals.render_texture_size[0] / (float) globals.render_texture_size[1];
    float fov_fac = tan(camera.fov / 2);
    float cone_radius = length(vec2(
            (2.0f * aspect_ratio * fov_fac) / (float) depth_texture.size[0],
            (2.0f * fov_fac) / (float) depth_texture.size[1]
    ));

    RayMarchHit hit = cone_march(ray, cone_radius, globals.time, DepthTextureEntry { 0.0, 0 });
    depth_texture.texture[id] = DepthTextureEntry { hit.depth, hit.steps };
}

extern "C" __global__ void render(char *render_texture, DepthTexture depth_texture, GlobalsBuffer globals, CameraBuffer camera)
{
    u32 id = blockIdx.x * blockDim.x + threadIdx.x;
    uvec2 texture_coord = uvec2(id % globals.render_texture_size[0], id / globals.render_texture_size[0]);
    vec2 ndc_coord = texture_to_ndc(texture_coord, {globals.render_texture_size[0], globals.render_texture_size[1] });
    vec2 cam_coord = ndc_to_camera(ndc_coord, { globals.render_texture_size[0], globals.render_texture_size[1] });
    Ray ray { { camera.position[0], camera.position[1], camera.position[2] }, camera_to_ray(cam_coord, camera) };

    DepthTextureEntry entry = depth_texture.texture[texture_coord.x / 8 + depth_texture.size[0] * (texture_coord.y / 8)];

    vec3 color = render_ray(ray, globals.time, entry);
    vec3 mapped_color = clamp(color, 0.0f, 1.0f);

    unsigned int rgba = ((unsigned int)(255.0f * mapped_color.x) & 0xff) |
                        (((unsigned int)(255.0f * mapped_color.y) & 0xff) << 8) |
                        (((unsigned int)(255.0f * mapped_color.z) & 0xff) << 16) |
                        (255 << 24);
    ((unsigned int*)render_texture)[id] = rgba;

    /*auto intensity = (unsigned int) (
        255.0 * clamp(depth_texture.texture[texture_coord.x / 16 + depth_texture.size[0] * (texture_coord.y / 16)] / 100.0, 0.0, 1.0)
    );
    unsigned int rgba = (intensity) | (intensity << 8) | (intensity << 16) | (255 << 24);
    ((unsigned int*)render_texture)[id] = rgba;*/
}
